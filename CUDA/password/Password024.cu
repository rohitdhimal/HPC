#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int cracks(char * passcrack){

   char pswd1[]="RD1234";
   char pswd2[]="MN4567";
   char pswd3[]="SS9867";
   char pswd4[]="CD5493";

   char *m1 = passcrack;
   char *m2 = passcrack;
   char *m3 = passcrack;
   char *m4 = passcrack;

   char *p1 = pswd1;
   char *p2 = pswd2;
   char *p3 = pswd3;
   char *p4 = pswd4;

   while(*m1 == *p1){
    
       if(*m1 == '\0'){
          return 1;
       }
       m1++;
       p1++;
   }
   
    while(*m2 == *p2){
    
       if(*m2 == '\0'){
          return 1;
       }
       m2++;
       p2++;
   }

    while(*m3 == *p3){
    
       if(*m3 == '\0'){
          return 1;
       }
       m3++;
       p3++;
   }

    while(*m4 == *p4){
    
       if(*m4 == '\0'){
          return 1;
       }
       m4++;
       p4++;
   }
 return 0;
}

__global__ void cracks() {

 
  char word[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  

  char numb[10] = {'0','1','2','3','4','5','6','7','8','9'};
  

  char passcrack[7];
  passcrack[6] = '\0';
  int m, n, o, p;

     for(m=0;m<10;m++){
      for(n=0; n<10; n++){
       for(o=0; o<10; o++){
        for(p=0; p<10; p++){

        passcrack[0] = word[blockIdx.x];
        passcrack[1] = word[threadIdx.x];
        passcrack[2] = numb[m];
        passcrack[3] = numb[n];
        passcrack[4] = numb[o];
        passcrack[5] = numb[p];

        if(cracks(passcrack)){
            printf("Password successfully cracked: %s\n", passcrack);
        }

       }
      }
     }
    }
   }

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
     long long int ds =  finish->tv_sec - start->tv_sec;
     long long int dn =  finish->tv_nsec - start->tv_nsec;

     if(dn < 0 )
     {
      ds--;
      dn += 1000000000;
      }

     *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}


int main(int argc, char *argv[])
{

    struct timespec start, finish;  
    long long int time_elapsed;

    clock_gettime(CLOCK_MONOTONIC, &start);

    cracks <<<26, 26>>>();

    hipDeviceSynchronize();


    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
     printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                           (time_elapsed/1.0e9));
  return 0;
}







